#include "../include/device_resource_manager.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

namespace deviceResourceManager {
namespace {
sudoku::Board::FieldValue *d_old_boards = nullptr;
int *d_old_boards_count = nullptr;
sudoku::Board::FieldValue *d_new_boards = nullptr;
int *d_new_boards_count = nullptr;
sudoku::Board::FieldValue *d_solved_board = nullptr;
int *d_solved_board_mutex = nullptr;
uint8_t *d_empty_fields = nullptr;
uint8_t *d_empty_fields_count = nullptr;

#define gpuErrchk(ans)                                                         \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}
} // namespace
sudoku::Board::FieldValue *GetOldBoards() {
  if (!d_old_boards)
    gpuErrchk(hipMalloc(reinterpret_cast<void **>(&d_old_boards),
                         kNBoards * sudoku::Board::kBoardSize *
                             sudoku::Board::kBoardSize *
                             sizeof(sudoku::Board::FieldValue)));

  return d_old_boards;
}

int *GetOldBoardsCount() {
  if (!d_old_boards_count)
    hipMalloc(reinterpret_cast<void **>(&d_old_boards_count), sizeof(int));
  return d_old_boards_count;
}

sudoku::Board::FieldValue *GetNewBoards() {
  if (!d_new_boards)
    gpuErrchk(hipMalloc(reinterpret_cast<void **>(&d_new_boards),
                         kNBoards * sudoku::Board::kBoardSize *
                             sudoku::Board::kBoardSize *
                             sizeof(sudoku::Board::FieldValue)));
  return d_new_boards;
}

int *GetNewBoardsCount() {
  if (!d_new_boards_count)
    gpuErrchk(hipMalloc(reinterpret_cast<void **>(&d_new_boards_count),
                         sizeof(int)));
  return d_new_boards_count;
}

sudoku::Board::FieldValue *GetSolvedBoard() {
  if (!d_solved_board)
    gpuErrchk(hipMalloc(reinterpret_cast<void **>(&d_solved_board),
                         sudoku::Board::kBoardSize * sudoku::Board::kBoardSize *
                             sizeof(sudoku::Board::FieldValue)));
  return d_solved_board;
}

int *GetSolvedBoardMutex() {
  if (!d_solved_board_mutex)
    gpuErrchk(hipMalloc(reinterpret_cast<void **>(&d_solved_board_mutex),
                         sizeof(int)));
  return d_solved_board_mutex;
}

uint8_t *GetEmptyFields() {
  if (!d_empty_fields)
    gpuErrchk(hipMalloc(reinterpret_cast<void **>(&d_empty_fields),
                         kNBoards * sudoku::Board::kBoardSize *
                             sudoku::Board::kBoardSize * sizeof(uint8_t)));
  return d_empty_fields;
}

uint8_t *GetEmptyFieldsCount() {
  if (!d_empty_fields_count)
    gpuErrchk(hipMalloc(reinterpret_cast<void **>(&d_empty_fields_count),
                         kNBoards * sizeof(uint8_t)));
  return d_empty_fields_count;
}

void Release() {
  hipFree(d_empty_fields_count);
  hipFree(d_empty_fields);
  hipFree(d_solved_board_mutex);
  hipFree(d_solved_board);
  hipFree(d_new_boards_count);
  hipFree(d_new_boards);
  hipFree(d_old_boards_count);
  hipFree(d_old_boards);
}
} // namespace deviceResourceManager
