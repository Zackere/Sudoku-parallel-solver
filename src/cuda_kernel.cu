#include "hip/hip_runtime.h"
#include "../include/cuda_kernel.cuh"

#include <cassert>
#include <cstdio>
#include <ctime>
#include <memory>
#include <utility>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>

namespace sudoku {
namespace kernel {
namespace {
constexpr unsigned kBlocks = 1024;
constexpr unsigned kThreadsPerBlock = 256;
constexpr unsigned kNBoards = 1 << 22;
constexpr unsigned kIterations = 9;

#define SetNthBit(number, n) ((number) |= (1ul << (n)))
#define ClearNthBit(number, n) ((number) &= ~(1ul << (n)))
#define GetNthBit(number, n) (((number) >> (n)) & 1u)

__device__ uint16_t GetPossibleValues(Board::FieldValue *board, int cell) {
  int row = cell / Board::kBoardSize;
  int col = cell - row * Board::kBoardSize;
  uint16_t free = 0x01ff;

  for (int i = 0; i < Board::kBoardSize; ++i) {
    auto val = board[row * Board::kBoardSize + i] - 1;
    if (val > -1)
      ClearNthBit(free, val);
    val = board[i * Board::kBoardSize + col] - 1;
    if (val > -1)
      ClearNthBit(free, val);
  }
  auto pom_y = row - row % Board::kQuadrantSize;
  auto pom_x = col - col % Board::kQuadrantSize;
  for (int i = 0; i < Board::kQuadrantSize; ++i)
    for (int j = 0; j < Board::kQuadrantSize; ++j) {
      auto val = board[(pom_y + i) * Board::kBoardSize + pom_x + j] - 1;
      if (val > -1)
        ClearNthBit(free, val);
    }
  return free;
}

__global__ void Generator(Board::FieldValue *old_boards, int *old_boards_count,
                          Board::FieldValue *new_boards, int *new_boards_count,
                          unsigned char *empty_fields,
                          unsigned char *empty_fields_count,
                          Board::FieldValue *solved_board,
                          int *solved_board_mutex) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x;
       index < *old_boards_count; index += blockDim.x * gridDim.x) {
    if (*solved_board_mutex)
      return;
    for (int i = index * Board::kBoardSize * Board::kBoardSize;
         i < (index + 1) * Board::kBoardSize * Board::kBoardSize; ++i) {
      if (!old_boards[i]) {
        auto pv = GetPossibleValues(
            old_boards + index * Board::kBoardSize * Board::kBoardSize,
            i - index * Board::kBoardSize * Board::kBoardSize);
        for (int j = 0; j < Board::kBoardSize; ++j) {
          if (GetNthBit(pv, j)) {
            auto pos = atomicAdd(new_boards_count, 1);
            if (pos < kNBoards) {
              old_boards[i] = j + 1;
              unsigned char empty_index = static_cast<unsigned char>(-1);
              for (int k = 0; k < Board::kBoardSize * Board::kBoardSize; ++k) {
                if (!(new_boards[pos * Board::kBoardSize * Board::kBoardSize +
                                 k] = old_boards
                          [index * Board::kBoardSize * Board::kBoardSize + k]))
                  empty_fields[++empty_index +
                               pos * Board::kBoardSize * Board::kBoardSize] =
                      pos * Board::kBoardSize * Board::kBoardSize + k;
              }
              empty_fields_count[pos] = empty_index + 1;
            }
          }
        }
        goto NOT_SOLVED;
      }
    }
    atomicCAS(solved_board_mutex, 0, blockIdx.x * blockDim.x + threadIdx.x);
    if (*solved_board_mutex != blockIdx.x * blockDim.x + threadIdx.x)
      return;
    for (int i = 0; i < Board::kBoardSize * Board::kBoardSize; ++i)
      solved_board[i] =
          old_boards[index * Board::kBoardSize * Board::kBoardSize + i];
  NOT_SOLVED:;
  }
}

__device__ bool NotInRow(Board::FieldValue *board, int row) {
  uint16_t st = 0;

  for (int i = 0; i < Board::kBoardSize; i++)
    if (board[Board::kBoardSize * row + i]) {
      if (GetNthBit(st, board[Board::kBoardSize * row + i]))
        return false;
      SetNthBit(st, board[Board::kBoardSize * row + i]);
    }
  return true;
}

__device__ bool NotInCol(Board::FieldValue *board, int col) {
  uint16_t st = 0;
  for (int i = 0; i < Board::kBoardSize; i++)
    if (board[Board::kBoardSize * i + col]) {
      if (GetNthBit(st, board[Board::kBoardSize * i + col]))
        return false;
      SetNthBit(st, board[Board::kBoardSize * i + col]);
    }
  return true;
}

__device__ bool NotInBox(Board::FieldValue *board, int startRow, int startCol) {
  uint16_t st = 0;
  for (int row = 0; row < Board::kQuadrantSize; ++row)
    for (int col = 0; col < Board::kQuadrantSize; ++col)
      if (board[Board::kBoardSize * row + col]) {
        if (GetNthBit(st, board[Board::kBoardSize * row + col]))
          return false;
        SetNthBit(st, board[Board::kBoardSize * row + col]);
      }
  return true;
}

__device__ bool IsValid(Board::FieldValue *board, int row, int col) {
  return NotInRow(board, row) && NotInCol(board, col) &&
         NotInBox(board, row - row % 3, col - col % 3);
}

__device__ bool IsValidConfig(Board::FieldValue *board) {
  for (int i = 0; i < Board::kBoardSize; ++i)
    for (int j = 0; j < Board::kBoardSize; ++j)
      if (!IsValid(board, i, j))
        return false;
  return true;
}

__device__ bool Solve(Board::FieldValue *board, unsigned char *empty_fields,
                      unsigned char empty_fields_count) {
  unsigned char empty_index = 0;
  while (empty_index < empty_fields_count) {
    ++board[empty_fields[empty_index]];
    if (!IsValidConfig(board)) {
      if (board[empty_fields[empty_index]] >= 9) {
        board[empty_fields[empty_index]] = 0;
        --empty_index;
      }
    } else {
      ++empty_index;
    }
  }
  return empty_fields_count == empty_index;
}

__global__ void Backtracker(Board::FieldValue *old_boards,
                            int *old_boards_count, unsigned char *empty_fields,
                            unsigned char *empty_fields_count,
                            Board::FieldValue *solved_board,
                            int *solved_board_mutex) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x;
       index < *old_boards_count; index += blockDim.x * gridDim.x) {
    if (*solved_board_mutex)
      return;
    if (Solve(old_boards + index * Board::kBoardSize * Board::kBoardSize,
              empty_fields + index * Board::kBoardSize * Board::kBoardSize,
              empty_fields_count[index])) {
      atomicCAS(solved_board_mutex, 0, blockIdx.x * blockDim.x + threadIdx.x);
      if (*solved_board_mutex != blockIdx.x * blockDim.x + threadIdx.x)
        return;
      for (int i = 0; i < Board::kBoardSize * Board::kBoardSize; ++i)
        solved_board[i] =
            old_boards[index * Board::kBoardSize * Board::kBoardSize + i];
    }
  }
}
} // namespace

std::vector<Board::FieldValue>
Run(std::vector<Board::FieldValue> const &board) {
  Board::FieldValue *d_old_boards = nullptr;
  hipMalloc(reinterpret_cast<void **>(&d_old_boards),
             kNBoards * Board::kBoardSize * Board::kBoardSize *
                 sizeof(Board::FieldValue));
  int *d_old_boards_count;
  hipMalloc(reinterpret_cast<void **>(&d_old_boards_count), sizeof(int));
  Board::FieldValue *d_new_boards = nullptr;
  hipMalloc(reinterpret_cast<void **>(&d_new_boards),
             kNBoards * Board::kBoardSize * Board::kBoardSize *
                 sizeof(Board::FieldValue));
  int *d_new_boards_count = nullptr;
  hipMalloc(reinterpret_cast<void **>(&d_new_boards_count), sizeof(int));

  hipMemset(d_old_boards, 0,
             kNBoards * Board::kBoardSize * Board::kBoardSize *
                 sizeof(Board::FieldValue));
  hipMemset(d_new_boards, 0,
             kNBoards * Board::kBoardSize * Board::kBoardSize *
                 sizeof(Board::FieldValue));
  std::unique_ptr<int> one(new int(1));
  hipMemcpy(d_old_boards_count, one.get(), sizeof(int),
             hipMemcpyHostToDevice);
  hipMemcpy(d_old_boards, board.data(),
             Board::kBoardSize * Board::kBoardSize * sizeof(Board::FieldValue),
             hipMemcpyHostToDevice);

  Board::FieldValue *d_solved_board = nullptr;
  hipMalloc(reinterpret_cast<void **>(&d_solved_board),
             Board::kBoardSize * Board::kBoardSize * sizeof(Board::FieldValue));
  hipMemset(d_solved_board, 0,
             Board::kBoardSize * Board::kBoardSize * sizeof(Board::FieldValue));

  int *d_solved_board_mutex = nullptr;
  hipMalloc(reinterpret_cast<void **>(&d_solved_board_mutex), sizeof(int));
  hipMemset(d_solved_board_mutex, 0, sizeof(int));

  unsigned char *d_empty_fields = nullptr;
  hipMalloc(reinterpret_cast<void **>(&d_empty_fields),
             kNBoards * Board::kBoardSize * Board::kBoardSize *
                 sizeof(unsigned char));
  hipMemset(d_empty_fields, 0,
             kNBoards * Board::kBoardSize * Board::kBoardSize *
                 sizeof(unsigned char));

  unsigned char *d_empty_fields_count = nullptr;
  hipMalloc(reinterpret_cast<void **>(&d_empty_fields_count),
             kNBoards * sizeof(unsigned char));
  hipMemset(d_empty_fields_count, 0, kNBoards * sizeof(unsigned char));

  for (int i = 0; i < kIterations; ++i) {
    hipMemset(d_new_boards_count, 0, sizeof(int));
    Generator<<<kBlocks, kThreadsPerBlock>>>(
        d_old_boards, d_old_boards_count, d_new_boards, d_new_boards_count,
        d_empty_fields, d_empty_fields_count, d_solved_board,
        d_solved_board_mutex);
    hipDeviceSynchronize();
    hipMemset(d_old_boards_count, 0, sizeof(int));
    Generator<<<kBlocks, kThreadsPerBlock>>>(
        d_new_boards, d_new_boards_count, d_old_boards, d_old_boards_count,
        d_empty_fields, d_empty_fields_count, d_solved_board,
        d_solved_board_mutex);
    hipDeviceSynchronize();
  }

  int solved = 0;
  hipMemcpy(&solved, d_solved_board_mutex, sizeof(int),
             hipMemcpyDeviceToHost);
  if (!solved)
    Backtracker<<<kBlocks, kThreadsPerBlock>>>(
        d_old_boards, d_old_boards_count, d_empty_fields, d_empty_fields_count,
        d_solved_board, d_solved_board_mutex);
  std::unique_ptr<Board::FieldValue[]> ret(
      new Board::FieldValue[Board::kBoardSize * Board::kBoardSize]);
  hipMemcpy(ret.get(), d_solved_board,
             Board::kBoardSize * Board::kBoardSize * sizeof(Board::FieldValue),
             hipMemcpyDeviceToHost);

  hipFree(d_empty_fields_count);
  hipFree(d_empty_fields);
  hipFree(d_solved_board_mutex);
  hipFree(d_solved_board);
  hipFree(d_new_boards_count);
  hipFree(d_new_boards);
  hipFree(d_old_boards_count);
  hipFree(d_old_boards);
  return {ret.get(), ret.get() + Board::kBoardSize * Board::kBoardSize};
}
} // namespace kernel
} // namespace sudoku
