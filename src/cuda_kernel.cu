#include "hip/hip_runtime.h"
#include "../include/cuda_kernel.cuh"

#include "../include/device_resource_manager.cuh"
#include "../include/time_manager.hpp"

#include <cassert>
#include <cstdio>
#include <ctime>
#include <memory>
#include <utility>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

namespace sudoku {
namespace kernel {
namespace {
constexpr unsigned kBlocks = 512;
constexpr unsigned kThreadsPerBlock = 128;
constexpr unsigned kIterations = 13;

#define SetNthBit(number, n) ((number) |= (1ul << (n)))
#define ClearNthBit(number, n) ((number) &= ~(1ul << (n)))
#define GetNthBit(number, n) (((number) >> (n)) & 1u)

__device__ bool NotInRow(Board::FieldValue *board, int row) {
  uint16_t st = 0;
  bool ret = true;
  Board::FieldValue v;
  for (int i = 0; i < Board::kBoardSize; i++) {
    v = board[Board::kBoardSize * row + i];
    ret = v <= Board::kBoardSize && ret;
    ret = !(v && GetNthBit(st, v)) && ret;
    SetNthBit(st, v);
  }
  return ret;
}

__device__ bool NotInCol(Board::FieldValue *board, int col) {
  uint16_t st = 0;
  bool ret = true;
  Board::FieldValue v;
  for (int i = 0; i < Board::kBoardSize; i++) {
    v = board[Board::kBoardSize * i + col];
    ret = v <= Board::kBoardSize && ret;
    ret = !(v && GetNthBit(st, v)) && ret;
    SetNthBit(st, v);
  }
  return ret;
}

__device__ bool NotInBox(Board::FieldValue *board, int row, int col) {
  row -= row % Board::kQuadrantSize;
  col -= col % Board::kQuadrantSize;
  uint16_t st = 0;
  bool ret = true;
  auto pom_y = row - row % Board::kQuadrantSize;
  auto pom_x = col - col % Board::kQuadrantSize;
  Board::FieldValue v;
  for (int i = 0; i < Board::kQuadrantSize; ++i)
    for (int j = 0; j < Board::kQuadrantSize; ++j) {
      v = board[(pom_y + i) * Board::kBoardSize + pom_x + j];
      ret = v <= Board::kBoardSize && ret;
      ret = !(v && GetNthBit(st, v)) && ret;
      SetNthBit(st, v);
    }
  return ret;
}

__device__ bool IsValid(Board::FieldValue *board, int row, int col) {
  return NotInRow(board, row) && NotInCol(board, col) &&
         NotInBox(board, row, col);
}

__global__ void Generator(Board::FieldValue *old_boards, int *old_boards_count,
                          Board::FieldValue *new_boards, int *new_boards_count,
                          unsigned char *empty_fields,
                          unsigned char *empty_fields_count,
                          Board::FieldValue *solved_board,
                          int *solved_board_mutex) {
  __shared__ Board::FieldValue s_current_boards[kThreadsPerBlock *
                                                Board::kBoardSize *
                                                Board::kBoardSize];
  auto *my_board =
      s_current_boards + threadIdx.x * Board::kBoardSize * Board::kBoardSize;
  for (int index = blockIdx.x * blockDim.x; index < *old_boards_count;
       index += blockDim.x * gridDim.x) {
    __syncthreads();
    if (*solved_board_mutex)
      return;
    for (int i = 0; i < Board::kBoardSize * Board::kBoardSize; ++i) {
      auto j = i * kThreadsPerBlock +
               index * Board::kBoardSize * Board::kBoardSize + threadIdx.x;
      s_current_boards[i * kThreadsPerBlock + threadIdx.x] =
          j < *old_boards_count * Board::kBoardSize * Board::kBoardSize
              ? old_boards[j]
              : 1;
    }
    __syncthreads();
    for (int i = 0; i < Board::kBoardSize * Board::kBoardSize; ++i) {
      if (!my_board[i]) {
        auto row = i / Board::kBoardSize;
        auto col = i % Board::kBoardSize;
        for (int j = 1; j <= Board::kBoardSize; ++j) {
          my_board[i] = j;
          if (IsValid(my_board, row, col)) {
            auto pos = atomicAdd(new_boards_count, 1);
            if (pos < deviceResourceManager::kNBoards) {
              unsigned char empty_index = static_cast<unsigned char>(-1);
              for (int k = 0; k < Board::kBoardSize * Board::kBoardSize; ++k) {
                if (!(new_boards[pos * Board::kBoardSize * Board::kBoardSize +
                                 k] = my_board[k]))
                  empty_fields[++empty_index +
                               pos * Board::kBoardSize * Board::kBoardSize] = k;
              }
              empty_fields_count[pos] = empty_index + 1;
            } else {
              atomicMin(new_boards_count, deviceResourceManager::kNBoards);
              return;
            }
          }
        }
        goto NOT_SOLVED;
      }
    }
    if (threadIdx.x + index < *old_boards_count) {
      atomicCAS(solved_board_mutex, 0, blockIdx.x * blockDim.x + threadIdx.x);
      if (*solved_board_mutex == blockIdx.x * blockDim.x + threadIdx.x)
        for (int i = 0; i < Board::kBoardSize * Board::kBoardSize; ++i)
          solved_board[i] = my_board[i];
    }
  NOT_SOLVED:;
  }
}

__device__ bool Solve(Board::FieldValue *board, uint8_t *empty_fields,
                      uint8_t empty_fields_count) {
  unsigned char empty_index = 0;
  auto field = empty_fields[empty_index];
  auto row = field / Board::kBoardSize;
  auto col = field % Board::kBoardSize;
  while (empty_index < empty_fields_count) {
    ++board[field];
    if (IsValid(board, row, col)) {
      field = empty_fields[++empty_index];
      row = field / Board::kBoardSize;
      col = field % Board::kBoardSize;
    } else {
      if (board[field] >= Board::kBoardSize) {
        board[field] = 0;
        field = empty_fields[--empty_index];
        row = field / Board::kBoardSize;
        col = field % Board::kBoardSize;
      }
    }
  }
  return empty_index == empty_fields_count;
}

__global__ void Backtracker(Board::FieldValue *old_boards,
                            int *old_boards_count, uint8_t *empty_fields,
                            uint8_t *empty_fields_count,
                            Board::FieldValue *solved_board,
                            int *solved_board_mutex) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x;
       index < *old_boards_count; index += blockDim.x * gridDim.x) {
    if (*solved_board_mutex)
      return;
    auto index_mul = index * Board::kBoardSize * Board::kBoardSize;
    if (Solve(old_boards + index_mul, empty_fields + index_mul,
              empty_fields_count[index])) {
      atomicCAS(solved_board_mutex, 0, blockIdx.x * blockDim.x + threadIdx.x);
      if (*solved_board_mutex != blockIdx.x * blockDim.x + threadIdx.x)
        return;
      for (int i = 0; i < Board::kBoardSize * Board::kBoardSize; ++i)
        solved_board[i] = old_boards[index_mul + i];
    }
  }
}

__device__ uint16_t GetPossibleValues(
    Board::FieldValue board[Board::kBoardSize][Board::kBoardSize]) {
  uint16_t free = 0x03ff;
  for (int i = 0; i < Board::kBoardSize; ++i) {
    ClearNthBit(free, board[threadIdx.y][i]);
    ClearNthBit(free, board[i][threadIdx.x]);
  }
  auto pom_x = threadIdx.x - threadIdx.x % Board::kQuadrantSize;
  auto pom_y = threadIdx.y - threadIdx.y % Board::kQuadrantSize;
  for (int i = 0; i < Board::kQuadrantSize; ++i)
    for (int j = 0; j < Board::kQuadrantSize; ++j)
      ClearNthBit(free,
                  board[pom_y + (threadIdx.y + j) % Board::kQuadrantSize]
                       [pom_x + (threadIdx.x + i) % Board::kQuadrantSize]);
  return free >> 1;
}

__global__ void Simplificator(Board::FieldValue *old_boards,
                              int *old_boards_count,
                              Board::FieldValue *new_boards,
                              int *new_boards_count) {
  __shared__ Board::FieldValue s_board[Board::kBoardSize][Board::kBoardSize];
  __shared__ int pos;
  pos = 0;
  for (int index = blockIdx.x; index < *old_boards_count; index += gridDim.x) {
    bool active =
        !(s_board[threadIdx.y][threadIdx.x] =
              (old_boards +
               index * Board::kBoardSize *
                   Board::kBoardSize)[Board::kBoardSize * threadIdx.y +
                                      threadIdx.x]);
    __syncthreads();
    auto pv = GetPossibleValues(s_board);
    auto nelems = __popc(pv);
    while (__syncthreads_or(active && nelems == 1)) {
      if (active && nelems == 1) {
        s_board[threadIdx.y][threadIdx.x] = __ffs(pv);
        active = false;
      }
      __syncthreads();
      if (active) {
        pv = GetPossibleValues(s_board);
        nelems = __popc(pv);
      }
    }
    if (__syncthreads_or(active && nelems == 0))
      continue;
    if (__syncthreads_and(
            IsValid(reinterpret_cast<Board::FieldValue *>(s_board), threadIdx.y,
                    threadIdx.x))) {
      if (threadIdx.x + threadIdx.y == 0)
        pos = atomicAdd(new_boards_count, 1);
      __syncthreads();
      (new_boards +
       pos * Board::kBoardSize *
           Board::kBoardSize)[Board::kBoardSize * threadIdx.y + threadIdx.x] =
          s_board[threadIdx.y][threadIdx.x];
    }
  }
}

class ScopedCudaEvent {
public:
  ScopedCudaEvent() { hipEventCreate(&event_); }
  ~ScopedCudaEvent() { hipEventDestroy(event_); }
  hipEvent_t Get() { return event_; }
  void Record() { hipEventRecord(event_); }
  void Sync() { hipEventSynchronize(event_); }

private:
  hipEvent_t event_;

  ScopedCudaEvent(ScopedCudaEvent const &) = delete;
  ScopedCudaEvent &operator=(ScopedCudaEvent const &) = delete;
};

class ScopedCudaStream {
public:
  ScopedCudaStream() { hipStreamCreate(&stream_); }
  ~ScopedCudaStream() { hipStreamDestroy(stream_); }
  hipStream_t Get() { return stream_; }
  hipError_t Query() { return hipStreamQuery(stream_); }
  void Sync() { hipStreamSynchronize(stream_); }

private:
  hipStream_t stream_;

  ScopedCudaStream(ScopedCudaStream const &) = delete;
  ScopedCudaStream &operator=(ScopedCudaStream const &) = delete;
};
} // namespace

std::vector<Board::FieldValue>
Run(std::vector<Board::FieldValue> const &board) {
  Board::FieldValue *d_old_boards = deviceResourceManager::GetOldBoards();
  int *d_old_boards_count = deviceResourceManager::GetOldBoardsCount();
  Board::FieldValue *d_new_boards = deviceResourceManager::GetNewBoards();
  int *d_new_boards_count = deviceResourceManager::GetNewBoardsCount();
  Board::FieldValue *d_solved_board = deviceResourceManager::GetSolvedBoard();
  int *d_solved_board_mutex = deviceResourceManager::GetSolvedBoardMutex();
  uint8_t *d_empty_fields = deviceResourceManager::GetEmptyFields();
  uint8_t *d_empty_fields_count = deviceResourceManager::GetEmptyFieldsCount();
  ScopedCudaStream kernel_stream;
  ScopedCudaStream old_boards_set_stream, new_boards_set_stream,
      empty_fields_set_stream, empty_fields_count_set_stream;

  ScopedCudaEvent start, stop;
  start.Record();

  hipMemsetAsync(d_old_boards, 0,
                  deviceResourceManager::kNBoards * Board::kBoardSize *
                      Board::kBoardSize * sizeof(Board::FieldValue),
                  old_boards_set_stream.Get());
  hipMemsetAsync(d_new_boards, 0,
                  deviceResourceManager::kNBoards * Board::kBoardSize *
                      Board::kBoardSize * sizeof(Board::FieldValue),
                  new_boards_set_stream.Get());
  hipMemsetAsync(d_empty_fields, 0,
                  deviceResourceManager::kNBoards * Board::kBoardSize *
                      Board::kBoardSize * sizeof(uint8_t),
                  empty_fields_set_stream.Get());
  hipMemsetAsync(d_empty_fields_count, 0,
                  deviceResourceManager::kNBoards * sizeof(uint8_t),
                  empty_fields_count_set_stream.Get());
  std::unique_ptr<int> one(new int(1));
  hipMemcpy(d_old_boards_count, one.get(), sizeof(int),
             hipMemcpyHostToDevice);
  hipMemcpy(d_old_boards, board.data(),
             Board::kBoardSize * Board::kBoardSize * sizeof(Board::FieldValue),
             hipMemcpyHostToDevice);
  hipMemset(d_solved_board, 0,
             Board::kBoardSize * Board::kBoardSize * sizeof(Board::FieldValue));
  hipMemset(d_solved_board_mutex, 0, sizeof(int));

  old_boards_set_stream.Sync();
  new_boards_set_stream.Sync();
  empty_fields_set_stream.Sync();
  empty_fields_count_set_stream.Sync();

  for (int i = 0; i < kIterations; ++i) {
    hipMemset(d_new_boards_count, 0, sizeof(int));
    Simplificator<<<kBlocks, dim3(Board::kBoardSize, Board::kBoardSize), 0,
                    kernel_stream.Get()>>>(d_old_boards, d_old_boards_count,
                                           d_new_boards, d_new_boards_count);
    kernel_stream.Sync();
    std::swap(d_old_boards, d_new_boards);
    std::swap(d_old_boards_count, d_new_boards_count);
    hipMemset(d_new_boards_count, 0, sizeof(int));
    Generator<<<kBlocks, kThreadsPerBlock, 0, kernel_stream.Get()>>>(
        d_old_boards, d_old_boards_count, d_new_boards, d_new_boards_count,
        d_empty_fields, d_empty_fields_count, d_solved_board,
        d_solved_board_mutex);
    kernel_stream.Sync();
    std::swap(d_old_boards, d_new_boards);
    std::swap(d_old_boards_count, d_new_boards_count);
    hipMemset(d_new_boards_count, 0, sizeof(int));
    Simplificator<<<kBlocks, dim3(Board::kBoardSize, Board::kBoardSize), 0,
                    kernel_stream.Get()>>>(d_old_boards, d_old_boards_count,
                                           d_new_boards, d_new_boards_count);
    kernel_stream.Sync();
    std::swap(d_old_boards, d_new_boards);
    std::swap(d_old_boards_count, d_new_boards_count);
    hipMemset(d_new_boards_count, 0, sizeof(int));
    Generator<<<kBlocks, kThreadsPerBlock, 0, kernel_stream.Get()>>>(
        d_old_boards, d_old_boards_count, d_new_boards, d_new_boards_count,
        d_empty_fields, d_empty_fields_count, d_solved_board,
        d_solved_board_mutex);
    kernel_stream.Sync();
    std::swap(d_old_boards, d_new_boards);
    std::swap(d_old_boards_count, d_new_boards_count);
  }

  int solved = 0;
  hipMemcpy(&solved, d_solved_board_mutex, sizeof(int),
             hipMemcpyDeviceToHost);
  if (!solved) {
    Backtracker<<<kBlocks, kThreadsPerBlock, 0, kernel_stream.Get()>>>(
        d_old_boards, d_old_boards_count, d_empty_fields, d_empty_fields_count,
        d_solved_board, d_solved_board_mutex);
    kernel_stream.Sync();
  }
  std::unique_ptr<Board::FieldValue[]> ret(
      new Board::FieldValue[Board::kBoardSize * Board::kBoardSize]);
  hipMemcpy(ret.get(), d_solved_board,
             Board::kBoardSize * Board::kBoardSize * sizeof(Board::FieldValue),
             hipMemcpyDeviceToHost);

  stop.Record();
  stop.Sync();
  float ms = 0;
  hipEventElapsedTime(&ms, start.Get(), stop.Get());
  timeManager::AddTimeElapsed(ms);

  return {ret.get(), ret.get() + Board::kBoardSize * Board::kBoardSize};
}
} // namespace kernel
} // namespace sudoku
